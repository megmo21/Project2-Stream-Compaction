#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
	thrust::device_vector<int> dv_in(idata, idata + n);
	thrust::device_vector<int> dv_out(odata, odata + n);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	//printf("%f - ", milliseconds);

	thrust::copy(dv_out.begin(), dv_out.end(), odata);

}

}
}
